#include "hip/hip_runtime.h"
// BFSTEST : Test breadth-first search in a graph.
// 
// example: cat sample.txt | ./bfstest 1
//
// John R. Gilbert, 17 Feb 20ll

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <sm_11_atomic_functions.h>
#include <hip/hip_runtime_api.h>
#include "cudaec.c"

#define cutilSafeCall(x) checkCudaErrors(x)
#define cutilCheckMsg(x) getLastCudaError(x)

/* global state */
struct timespec	start_time;																 
struct timespec	end_time;	

int nedges, maxv;

unsigned int seed = 0x12345678;
unsigned int myrand(unsigned int *seed, unsigned int input) {	
	*seed = (*seed << 13) ^ (*seed >> 15) + input + 0xa174de3;
	return *seed;
};

void sig_check(char *level, int nv) {		
	int i;
	unsigned int sig = 0x123456;
	
	for(i = 0; i < nv; i++)
	{		
		myrand(&sig, level[i]);		
	}					 
		
	printf("Computed check sum signature:0x%08x\n", sig);
	if(sig == 0x18169857)
		printf("Result check of sample.txt by signature successful!!\n");
	else if(sig == 0xef872cf0)
		printf("Result check of TEST1 by signature successful!!\n");
	else if(sig == 0xe61d1d00) 
		printf("Result check of TEST3 by signature successful!!\n");
	else if(sig == 0x29c12a44)
		printf("Result check of TEST2 by signature successful!!\n");
	else
		printf("Result check by signature failed!!\n");
}

/* Read input from stdio (for genx.pl files, no more than 40 seconds) */
void read_edge_list (int **tailp, int **headp) {
	int max_edges = 100000000;
	int nr, t, h;
	
	*tailp = (int *) calloc(max_edges, sizeof(int));
	*headp = (int *) calloc(max_edges, sizeof(int));
	nedges = 0;
	maxv = 0;
	nr = scanf("%i %i",&t,&h);
	while (nr == 2) {
		if (nedges >= max_edges) {
			printf("Limit of %d edges exceeded.\n",max_edges);
			exit(1);
		}
		
		if (t > maxv) maxv = t;
		if (h > maxv) maxv = h;
		
		(*tailp)[nedges] = t;
		(*headp)[nedges++] = h;
		nr = scanf("%i %i",&t,&h);
	}
}

__global__ void bfs_cuda(int vsize, char* d_frontier, char* d_visited, char* d_cost, int* d_firstnbr, int* d_nbr, int* d_over, int nlevels){
	const int nodeid = blockIdx.x * 1024 + threadIdx.x;
	int head, tail, temp, frontier_sel = nlevels%2;

	if (nodeid > vsize) return;
	if (frontier_sel == 1 && (d_frontier[nodeid] & 0x80) == 0) return;
	else if (frontier_sel == 0 && (d_frontier[nodeid] & 0x1) == 0) return;

	d_frontier[nodeid] = (frontier_sel == 1) ? (d_frontier[nodeid] & 0x7F) : (d_frontier[nodeid] & 0xFE);
	d_visited[nodeid] = 1;
	head = (nodeid == 0) ? 0 : d_firstnbr[nodeid-1];
	tail = d_firstnbr[nodeid];

	while (head != tail){
		temp = d_nbr[head];
		if (!d_visited[temp]){
			d_frontier[temp] = (frontier_sel == 1) ? 0x1 : 0x80;
			d_cost[temp] = d_cost[nodeid] + 1;
			*d_over = 1;
		}
		head++;
	}
};

__global__ void bfs_cuda_init(int vsize, int startvtx, char* d_frontier, char* d_visited, char* d_cost){
	const int nodeid = blockIdx.x * 1024 + threadIdx.x;
	if (nodeid > vsize) return;
	d_frontier[nodeid] = (nodeid == startvtx) ? 1 : 0;
	d_visited[nodeid] = 0;
	d_cost[nodeid] = (nodeid == startvtx) ? 0 : -1;
};

int main (int argc, char* argv[]) {
	int *head, *tail;
	int *h_firstnbr, *h_nbr;
	int *d_firstnbr, *d_nbr;
	char *h_cost;
	char *d_frontier, *d_visited, *d_cost;
	int block, grid, nlevels = 0;
	int startvtx;
	int i, j;
	int vsize;
	int h_over, *d_over;

	if (argc == 2) {
		startvtx = atoi (argv[1]);
	} else {
		printf("usage:	 bfstest <startvtx> < <edgelistfile>\n");
		printf("example: cat sample.txt | ./bfstest 1\n");
		exit(1);
	}
	
	read_edge_list (&tail, &head);

	clock_gettime(CLOCK_REALTIME, &start_time); //stdio scanf ended, timer starts	//Don't remove it

	vsize = maxv+1;
	block = (vsize > 1023) ? 1024 : vsize;
	grid = (vsize >> 10) + 1;
	
	h_nbr = (int *) calloc(nedges, sizeof(int));
	h_firstnbr = (int *) calloc(vsize+1, sizeof(int));
	h_cost = (char *) malloc(vsize * sizeof(char));

	// count neighbors of vertex v in firstnbr[v+1],
	for (i = 0; i < nedges; i++) h_firstnbr[tail[i]+1]++;

	// cumulative sum of neighbors gives firstnbr[] values
	for (i = 0; i < vsize; i++) h_firstnbr[i+1] += h_firstnbr[i];

	// pass through edges, slotting each one into the CSR structure
	for (i = 0; i < nedges; i++) {
		j = h_firstnbr[tail[i]]++;
		h_nbr[j] = head[i];
	}

	// Allocate vectors in device memory
	cutilSafeCall( hipMalloc((void**)&d_firstnbr, (vsize+1)*sizeof(int)) );
    CudaCheckError();
	cutilSafeCall( hipMalloc((void**)&d_nbr, nedges*sizeof(int)) );
    CudaCheckError();
	cutilSafeCall( hipMalloc((void**)&d_frontier, vsize*sizeof(char)) );
    CudaCheckError();
	cutilSafeCall( hipMalloc((void**)&d_visited, vsize*sizeof(char)) );
    CudaCheckError();
	cutilSafeCall( hipMalloc((void**)&d_cost, vsize*sizeof(char)) );
    CudaCheckError();
	cutilSafeCall( hipMalloc((void**)&d_over, sizeof(int)) );
    CudaCheckError();

	// Copy vectors from host memory to device memory
	cutilSafeCall( hipMemcpy(d_firstnbr, h_firstnbr, (vsize+1)*sizeof(int), hipMemcpyHostToDevice) );		
    CudaCheckError();
	cutilSafeCall( hipMemcpy(d_nbr, h_nbr, nedges*sizeof(int), hipMemcpyHostToDevice) );
    CudaCheckError();
	bfs_cuda_init<<<grid, block>>>(vsize, startvtx, d_frontier, d_visited, d_cost);
    CudaCheckError();

    while (1){
        h_over = 0;
        cutilSafeCall( hipMemcpy(d_over, &h_over, sizeof(int), hipMemcpyHostToDevice) );
        CudaCheckError();
        bfs_cuda<<<grid, block>>>(vsize, d_frontier, d_visited, d_cost, d_firstnbr, d_nbr, d_over, nlevels);
        cutilSafeCall( hipDeviceSynchronize() );
        CudaCheckError();
        nlevels++;
        cutilSafeCall( hipMemcpy(&h_over, d_over, sizeof(int), hipMemcpyDeviceToHost) );
        CudaCheckError();
        if (!h_over) break;
    }
		
	cutilSafeCall( hipMemcpy(h_cost, d_cost, vsize*sizeof(char), hipMemcpyDeviceToHost) );
    CudaCheckError();

	clock_gettime(CLOCK_REALTIME, &end_time);	//graph construction and bfs completed timer ends	//Don't remove it

	printf("Starting vertex for BFS is %d.\n\n",startvtx);
		
	//Don't remove it
	printf("s_time.tv_sec:%ld, s_time.tv_nsec:%09ld\n", start_time.tv_sec, start_time.tv_nsec);
	printf("e_time.tv_sec:%ld, e_time.tv_nsec:%09ld\n", end_time.tv_sec, end_time.tv_nsec);
	if(end_time.tv_nsec > start_time.tv_nsec)
	{
		printf("[diff_time:%ld.%09ld sec]\n",
		end_time.tv_sec - start_time.tv_sec,
		end_time.tv_nsec - start_time.tv_nsec);
	}
	else
	{
		printf("[diff_time:%ld.%09ld sec]\n",
		end_time.tv_sec - start_time.tv_sec - 1,
		end_time.tv_nsec - start_time.tv_nsec + 1000*1000*1000);
	}

	sig_check(h_cost, vsize);
	free(h_cost);
	free(h_firstnbr);
	free(h_nbr);
	free(tail);
	free(head);
}
